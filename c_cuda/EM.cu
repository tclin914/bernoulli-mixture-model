#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <float.h>
#include "EM.h"

int *imagesptr;
double *muptr;
double *piptr;
double *zptr;
int *kptr;
int *dptr;


__device__ double Cuda_ExpectationSubStep(int d_size, int n, int k, double *pi, double *mu, int *x) {
    double z_nk = pi[k];
    int i;
    for (i = 0; i < d_size; i++) {
        z_nk = z_nk + 
            pow(mu[d_size * k + i], x[d_size * n + i]) * 
            pow(1.0 - mu[d_size * k + i], 1.0 - x[d_size * n + i]);
    }
    return z_nk;
}

__global__ void Cuda_Expectation(int *k_size, int *d_size, int *image_width, double *pi, double *z, 
        double *mu, int *x) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    double normalizationFactor = 0.0;
    int k;
    for (k = 0; k < *k_size; k++) {
        z[*k_size * index + k] = Cuda_ExpectationSubStep(*d_size, index, k, pi, mu, x);
        normalizationFactor = normalizationFactor + z[*k_size * index + k];
    }

    for (k = 0; k < *k_size; k++) {
        if (normalizationFactor > 0.0) {
            z[(*k_size) * index + k] = z[(*k_size) * index + k] / normalizationFactor;
        } else {
            z[(*k_size) * index + k] = 1.0 / (float)(*k_size);
        }
    }
}

void EM(int *train_images, int *train_labels, int *test_images, int *test_labels, double *mu, double *pi, double *z) {

    srand(time(NULL));
    /* normalization */
    double normalizationFactor;
    int w, g;
    for (w = 0; w < K; w++) {
        normalizationFactor = 0;
        for (g = 0; g < D; g++) {
            mu[D * w + g] = rand() / (double)RAND_MAX;
            normalizationFactor = normalizationFactor + mu[D * w + g];
        }

        for (g = 0; g < D; g++) {
            mu[D * w + g] = mu[D * w + g] / normalizationFactor;
        }
    }

    hipMalloc((void**)&imagesptr, sizeof(int) * N * D);
    hipMalloc((void**)&muptr, sizeof(double) * K * D);
    hipMalloc((void**)&piptr, sizeof(double) * K);
    hipMalloc((void**)&zptr, sizeof(double) * N * K);

    hipMemcpy(imagesptr, train_images, sizeof(int) * N * D, hipMemcpyHostToDevice);
    hipMemcpy(muptr, mu, sizeof(double) * K * D, hipMemcpyHostToDevice);
    hipMemcpy(piptr, pi, sizeof(double) * K, hipMemcpyHostToDevice);
    hipMemcpy(zptr, z, sizeof(double) * N * K, hipMemcpyHostToDevice);
    
    int threadPerBlock = 500;
    int numBlocks = N / 500;



    /* for (int i = 0; i < 3; i++) { */
        /* ExpectationStep(z, pi, mu, train_images); */
        /* MaximizationStep(z, pi, mu, train_images);  */
    /* } */
}

void ExpectationStep(double z[N][K], double *pi, double mu[K][D], int **x) {
    double normalizationFactor;
    int n, k;
    for (n = 0; n < N; n++) {
        normalizationFactor = 0.0;

        for (k = 0; k < K; k++) {
            z[n][k] = ExpectationSubStep(n, k, pi, mu, x);
            normalizationFactor = normalizationFactor + z[n][k];
        }

        for (k = 0; k < K; k++) {
            if (normalizationFactor > 0.0) {
                z[n][k] = z[n][k] / normalizationFactor;
            } else {
                z[n][k] = 1.0 / (float)K;
            }
        }
    }
}

double ExpectationSubStep(int n, int k, double *pi, double mu[K][D], int **x) {
    double z_nk = pi[k];
    int i;
    for (i = 0; i < D; i++) {
        z_nk = z_nk * pow(mu[k][i], x[n][i]) * pow(1.0 - mu[k][i], 1.0 - x[n][i]);      
    }
    return z_nk;
}

void MaximizationStep(double z[N][K],double *pi, double mu[K][D], int **x) {
    int k, i;
    for (k = 0; k < K; k++) {
        pi[k] = Nm(k, z) / (double)N;
    } 
    double *average;
    for (k = 0; k < K; k++) {
        average = Average(k, x, z);
        
        for (i = 0; i < D; i++) {
            mu[k][i] = average[i];
        }
    }
    free(average);
}

double *Average(int m, int **x, double z[N][K]) {
    double *result = (double*)malloc(sizeof(double) * D);
    memset(result, 0, sizeof(double) * D);
    int i, n;
    for (i = 0; i < D; i++) {
        for (n = 0; n < N; n++) {
            result[i] = result[i] + z[n][m] * x[n][i];
        }
    }
    double currentNm = Nm(m, z);
    for (i = 0; i < D; i++) {
        result[i] = result[i] / currentNm;
    }
    return result;
}

double Nm(int m, double z[N][K]) {
    double result = 0.0;
    int n;
    for (n = 0; n < N; n++) {
        result = result + z[n][m];
    }
    return result;
}

int GetCluster(double mu[K][D], int *image) {
    double maxClusterSum = -DBL_MAX;
    int maxCluster = -1;
    int k, i;
    for (k = 0; k < K; k++) {
        double currentClusterSum = 0.0;
        for (i = 0; i < D; i++) {
            currentClusterSum += image[i] ? mu[k][i] : 1.0 - mu[k][i];
        }

        if (currentClusterSum > maxClusterSum) {
            maxClusterSum = currentClusterSum;
            maxCluster = k;
        }
    }
    return maxCluster;
}
