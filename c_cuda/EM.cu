#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <float.h>
#include "EM.h"

int *imagesptr;
double *muptr;
double *piptr;
double *zptr;
int *kptr;
int *dptr;
int *nptr;
int *resultptr;
int result[K * D];

__device__ double Cuda_ExpectationSubStep(int *d_size, int n, int k, double *pi, double *mu, int *x) {
    double z_nk = pi[k];
    int i;
    for (i = 0; i < *d_size; i++) {
        z_nk = z_nk * 
            pow(mu[(*d_size) * k + i], x[(*d_size) * n + i]) * 
            pow(1.0 - mu[(*d_size) * k + i], 1.0 - x[(*d_size) * n + i]);
    }
    return z_nk;
}

__global__ void Cuda_ExpectationStep(int *k_size, int *d_size, double *pi, double *z, double *mu, int *x) {
    int n = (blockIdx.x * blockDim.x) + threadIdx.x;
    double normalizationFactor = 0.0;
    int k;

    for (k = 0; k < *k_size; k++) {
        z[(*k_size) * n + k] = Cuda_ExpectationSubStep(d_size, n, k, pi, mu, x);
        normalizationFactor = normalizationFactor + z[(*k_size) * n + k];
    }

    for (k = 0; k < *k_size; k++) {
        if (normalizationFactor > 0.0) {
            z[(*k_size) * n + k] = z[(*k_size) * n + k] / normalizationFactor;
        } else {
            z[(*k_size) * n + k] = 1.0 / (float)(*k_size);
        }
    }
}

__device__ double Nm(int m, int *k_size, int *n_size, double *z) {
    double result = 0.0;
    int n;
    for (n = 0; n < *n_size; n++) {
        result = result + z[(*k_size) * n + m];
    }
    return result;
}

__device__ void Average(int m, int *k_size, int *d_size, int *n_size, 
        int *x, double *z, double *mu) {
    double currentNm = Nm(m, k_size, n_size, z);
    int i, n;
    for (i = 0; i < *d_size; i++) {
        double result = 0.0;
        for (n = 0; n < *n_size; n++) {
            result = result + z[(*k_size) * n + m] * x[(*d_size) * n + i];
        }
        mu[(*d_size) * m + i] = result / currentNm; 
    }
}

__global__ void Cuda_MaximizationStep(int *k_size, int *d_size, int *n_size, 
        double *pi, double *z, double *mu, int *x) {     
    int k = threadIdx.x;
    pi[k] = Nm(k, k_size, n_size, z) / (double)(*n_size);
    
    Average(k, k_size, d_size, n_size, x, z, mu);
}

__global__ void test(double *pi) {
    int i;
    for (i = 0; i < 40; i++) {
        pi[i] = 2.5;
    }
}

void EM(int *train_images, int *train_labels, int *test_images, int *test_labels, double *mu, double *pi, double *z) {

    srand(time(NULL));
    /* normalization */
    double normalizationFactor;
    int w, g;
    for (w = 0; w < K; w++) {
        normalizationFactor = 0;
        for (g = 0; g < D; g++) {
            mu[D * w + g] = rand() / (double)RAND_MAX;
            normalizationFactor = normalizationFactor + mu[D * w + g];
        }

        for (g = 0; g < D; g++) {
            mu[D * w + g] = mu[D * w + g] / normalizationFactor;
        }
    }
    
    int count;
    hipGetDeviceCount(&count);
    if (count == 0) {
        fprintf(stderr, "Here is no cuda device\n");
        exit(1);
    }

    int i;
    for (i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            
        }
    }
    hipSetDevice(i);

    hipMalloc((void**)&imagesptr, sizeof(int) * N * D);
    hipMalloc((void**)&muptr, sizeof(double) * K * D);
    hipMalloc((void**)&piptr, sizeof(double) * K);
    hipMalloc((void**)&zptr, sizeof(double) * N * K);
    hipMalloc((void**)&kptr, sizeof(int));
    hipMalloc((void**)&dptr, sizeof(int));
    hipMalloc((void**)&nptr, sizeof(int));

    int k_size = K;
    int d_size = D;
    int n_size = N;

    hipMemcpy(imagesptr, train_images, sizeof(int) * N * D, hipMemcpyHostToDevice);
    hipMemcpy(muptr, mu, sizeof(double) * K * D, hipMemcpyHostToDevice);
    hipMemcpy(piptr, pi, sizeof(double) * K, hipMemcpyHostToDevice);
    hipMemcpy(zptr, z, sizeof(double) * N * K, hipMemcpyHostToDevice);
    hipMemcpy(kptr, &k_size, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dptr, &d_size, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(nptr, &n_size, sizeof(int), hipMemcpyHostToDevice);


    int threadPerBlock, numBlocks;

    for (i = 0; i < 3; i++) {
        threadPerBlock = 500;
        numBlocks = N / 500;

        Cuda_ExpectationStep<<<numBlocks, threadPerBlock>>>(kptr, dptr, piptr, zptr, muptr, imagesptr);

        threadPerBlock = K;
        numBlocks = 1;

        Cuda_MaximizationStep<<<numBlocks, threadPerBlock>>>(kptr, dptr, nptr, piptr, zptr, muptr, imagesptr);
    }

    hipMemcpy(mu, muptr, sizeof(double) * K * D, hipMemcpyDeviceToHost);

    hipFree(imagesptr);
    hipFree(muptr);
    hipFree(piptr);
    hipFree(zptr);
    hipFree(kptr);
    hipFree(dptr);
    hipFree(nptr);
}

int GetCluster(double *mu, int *image) {
    double maxClusterSum = -DBL_MAX;
    int maxCluster = -1;
    int k, i;
    for (k = 0; k < K; k++) {
        double currentClusterSum = 0.0;
        for (i = 0; i < D; i++) {
            currentClusterSum += image[i] ? mu[D * k + i] : 1.0 - mu[D * k + i];
        }

        if (currentClusterSum > maxClusterSum) {
            maxClusterSum = currentClusterSum;
            maxCluster = k;
        }
    }
    return maxCluster;
}
